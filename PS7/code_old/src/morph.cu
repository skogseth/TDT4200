#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <cstring>
#include <string>
#include <cmath>
#include <pthread.h>

#include <chrono> // For timing

#ifdef __APPLE__
#  include <GLUT/glut.h>
#else
#  include <GL/glut.h>
#endif

using namespace std;

//the pixel
typedef struct pix{
  unsigned char r,g,b,a;
} pixel;

typedef struct SimplePoint_struct {
        float x, y;

} SimplePoint;


typedef struct SimpleFeatureLine_struct {
      SimplePoint startPoint;
        SimplePoint endPoint;

} SimpleFeatureLine;


template <typename T>
__host__ __device__ T CLAMP(T value, T low, T high)
{
        return (value < low) ? low : ((value > high) ? high : value);
}

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


int imgWidthOrig, imgHeightOrig, imgWidthDest, imgHeightDest;
int steps;
float p,a,b,t;
pixel* hSrcImgMap;
pixel* hDstImgMap;

//The name of input and output files
string inputFileSrc;
string inputFileDest;
string inputFileLines;
string outputPath;
string tempFile;
string dataPath;
string stepsStr;
string pStr, aStr, bStr, tStr;

void imgRead(string filename, pixel * &map, int &imgW, int &imgH){
    stbi_set_flip_vertically_on_load(true);

    int x, y, componentsPerPixel;
    if( ! filename.empty()  ){
        map = (pixel *) stbi_load(filename.c_str(), &x, &y, &componentsPerPixel, STBI_rgb_alpha);
    } else{
        cout<<"The input file name cannot be empty"<<endl;
        exit(1);
    }

    // get the current image columns and rows
    imgW = x;
    imgH = y;

    cout<<"Read the image file \""<<filename<<"\" successfully ."<<endl;
}

void imgWrite(string filename, pixel * map, int imgW, int imgH){
    if(filename.empty()){
        cout<<"The output file name cannot be empty"<<endl;
        exit(1);
    }
    stbi_flip_vertically_on_write(true);
    stbi_write_png(filename.c_str(), imgW, imgH, STBI_rgb_alpha, map, sizeof(pixel) * imgW);
    cout<<"Write the image into \""<<filename<<"\" file successfully."<<endl;
}

void loadLines(SimpleFeatureLine** linesSrc, SimpleFeatureLine** linesDst, int* linesLen, const char* name) {
	FILE *f = fopen(name, "r");
	if (f == NULL)
	{
		printf("Error opening file %s! \n", name);
		exit(1);
	}
	fscanf(f, "%d", linesLen);
	SimpleFeatureLine* srcLines = (SimpleFeatureLine*) malloc(sizeof(SimpleFeatureLine)*(*linesLen));
	SimpleFeatureLine* dstLines = (SimpleFeatureLine*) malloc(sizeof(SimpleFeatureLine)*(*linesLen));
	SimpleFeatureLine* line;
	int fac = 2;

	for(int i = 0; i < (*linesLen)*fac; i++) {
		line = (i % fac) ? &dstLines[(i-1)/fac] : &srcLines[i/fac];
		fscanf(f, "%f,%f,%f,%f",
				&(line->startPoint.x), &(line->startPoint.y),
				&(line->endPoint.x), &(line->endPoint.y));
	}

	*linesSrc = srcLines;
	*linesDst = dstLines;
}


// Parse commandline arguments
void parse(int argc, char *argv[]) {
    p = 0;
    a = 1;
    b = 2;
    t = 0.5;
    steps = 90;
    switch(argc){
        case 2:
            dataPath = argv[1];
        case 6:
            inputFileSrc = argv[1];
            inputFileDest = argv[2];
            inputFileLines = argv[3];
            outputPath = argv[4];
            stepsStr = argv[5];
            istringstream ( stepsStr ) >> steps;
            imgRead(inputFileSrc, hSrcImgMap, imgWidthOrig, imgHeightOrig);
            imgRead(inputFileDest, hDstImgMap, imgWidthDest, imgHeightDest);
            break;

        case 9:
            inputFileSrc = argv[1];
            inputFileDest = argv[2];
            inputFileLines = argv[3];
            outputPath = argv[4];
            stepsStr = argv[5];
            pStr = argv[6];
            aStr = argv[7];
            bStr = argv[8];
            istringstream ( stepsStr ) >> steps;
            istringstream ( pStr ) >> p;
            istringstream ( aStr ) >> a;
            istringstream ( bStr ) >> b;
            imgRead(inputFileSrc, hSrcImgMap, imgWidthOrig, imgHeightOrig);
            imgRead(inputFileDest, hDstImgMap, imgWidthDest, imgHeightDest);
            break;

        default:
            cout<<"Usage:"<<endl;
            cout<<"./morph srcImg.png destImg.png lines.txt outputPath steps [p] [a] [b]"<<endl;
            exit(1);
    }

}


void simpleLineInterpolate(SimpleFeatureLine* sourceLines,
                     SimpleFeatureLine* destLines , SimpleFeatureLine** morphLines, int linesLen, float t)
{
	SimpleFeatureLine* interLines = (SimpleFeatureLine*) malloc(sizeof(SimpleFeatureLine)*linesLen);
	for(int i=0; i<linesLen; i++){
		interLines[i].startPoint.x = (1-t)*(sourceLines[i].startPoint.x) + t*(destLines[i].startPoint.x);
		interLines[i].startPoint.y = (1-t)*(sourceLines[i].startPoint.y) + t*(destLines[i].startPoint.y);
		interLines[i].endPoint.x = (1-t)*(sourceLines[i].endPoint.x) + t*(destLines[i].endPoint.x);
		interLines[i].endPoint.y = (1-t)*(sourceLines[i].endPoint.y) + t*(destLines[i].endPoint.y);
	}
	*morphLines = interLines;
}




/* warping function (backward mapping)
   input:
   interPt = the point in the intermediary image
   interLines = given line in the intermediary image
   srcLines = given line in the source image
   p, a, b = parameters of the weight function
   output:
   src = the corresponding point */
__host__ __device__ void warp(const SimplePoint* interPt, SimpleFeatureLine* interLines,
          SimpleFeatureLine* sourceLines, const int sourceLinesSize, SimplePoint* src)
{
	int i;
	float interLength, srcLength;
	float weight, weightSum, dist;
	float sum_x, sum_y; // weighted sum of the coordination of the point "src"
	float u, v;
	SimplePoint pd, pq, qd;
	float X, Y;

	sum_x = 0;
	sum_y = 0;
	weightSum = 0;

	for (i=0; i<sourceLinesSize; i++) {
		pd.x = interPt->x - interLines[i].startPoint.x;
		pd.y = interPt->y - interLines[i].startPoint.y;
		pq.x = interLines[i].endPoint.x - interLines[i].startPoint.x;
		pq.y = interLines[i].endPoint.y - interLines[i].startPoint.y;
		interLength = pq.x * pq.x + pq.y * pq.y;
		u = (pd.x * pq.x + pd.y * pq.y) / interLength;

		interLength = sqrt(interLength); // length of the vector PQ

		v = (pd.x * pq.y - pd.y * pq.x) / interLength;

		pq.x = sourceLines[i].endPoint.x - sourceLines[i].startPoint.x;
		pq.y = sourceLines[i].endPoint.y - sourceLines[i].startPoint.y;

		srcLength = sqrt(pq.x * pq.x + pq.y * pq.y); // length of the vector P'Q'
		// corresponding point based on the ith line
		X = sourceLines[i].startPoint.x + u * pq.x + v * pq.y / srcLength;
		Y = sourceLines[i].startPoint.y + u * pq.y - v * pq.x / srcLength;

		// the distance from the corresponding point to the line P'Q'
		if (u < 0)
			dist = sqrt(pd.x * pd.x + pd.y * pd.y);
		else if (u > 1) {
			qd.x = interPt->x - interLines[i].endPoint.x;
			qd.y = interPt->y - interLines[i].endPoint.y;
			dist = sqrt(qd.x * qd.x + qd.y * qd.y);
		}else{
			dist = abs(v);
		}

		weight = pow(1.0f / (1.0f + dist), 2.0f);
		sum_x += X * weight;
		sum_y += Y * weight;
		weightSum += weight;
	}

	src->x = sum_x / weightSum;
	src->y = sum_y / weightSum;
}

__host__ __device__ void bilinear(pixel* Im, float row, float col, pixel* pix, int dImgWidth)
{
	int cm, cn, fm, fn;
	double alpha, beta;

	cm = (int)ceil(row);
	fm = (int)floor(row);
	cn = (int)ceil(col);
	fn = (int)floor(col);
	alpha = ceil(row) - row;
	beta = ceil(col) - col;

	pix->r = (unsigned int)( alpha*beta*Im[fm*dImgWidth+fn].r
			+ (1-alpha)*beta*Im[cm*dImgWidth+fn].r
			+ alpha*(1-beta)*Im[fm*dImgWidth+cn].r
			+ (1-alpha)*(1-beta)*Im[cm*dImgWidth+cn].r );
	pix->g = (unsigned int)( alpha*beta*Im[fm*dImgWidth+fn].g
			+ (1-alpha)*beta*Im[cm*dImgWidth+fn].g
			+ alpha*(1-beta)*Im[fm*dImgWidth+cn].g
			+ (1-alpha)*(1-beta)*Im[cm*dImgWidth+cn].g );
	pix->b = (unsigned int)( alpha*beta*Im[fm*dImgWidth+fn].b
			+ (1-alpha)*beta*Im[cm*dImgWidth+fn].b
			+ alpha*(1-beta)*Im[fm*dImgWidth+cn].b
			+ (1-alpha)*(1-beta)*Im[cm*dImgWidth+cn].b );
	pix->a = 255;
}

__host__ __device__ void ColorInterPolate(const SimplePoint* Src_P,
                      const SimplePoint* Dest_P, float t,
                      pixel* imgSrc, pixel* imgDest, pixel* rgb, int dImgWidth)
{
    pixel srcColor, destColor;

    bilinear(imgSrc, Src_P->y, Src_P->x, &srcColor, dImgWidth);
    bilinear(imgDest, Dest_P->y, Dest_P->x, &destColor, dImgWidth);

    rgb->b = srcColor.b*(1-t)+ destColor.b*t;
    rgb->g = srcColor.g*(1-t)+ destColor.g*t;
    rgb->r = srcColor.r*(1-t)+ destColor.r*t;
    rgb->a = 255;
}


///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////
///// DO NOT TOUCH CODE ABOVE. YOU ONLY NEED TO CHANGE THE FUNCTIONS BELOW ////////
///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

void morphKernel(SimpleFeatureLine* dSrcLines, SimpleFeatureLine* dDstLines, SimpleFeatureLine* dMorphLines,
		pixel* dSrcImgMap, pixel* dDstImgMap,  pixel* dMorphMap,
		int linesLen, int dImgWidth, int dImgHeight, float dT) {

	for (int i = 0; i < dImgHeight; i++) {
		for (int j = 0; j < dImgWidth; j++) {
			pixel interColor;
			SimplePoint dest;
			SimplePoint src;
			SimplePoint q;
			q.x = j;
			q.y = i;

			// warping
			warp(&q, dMorphLines, dSrcLines, linesLen, &src);
			warp(&q, dMorphLines, dDstLines, linesLen, &dest);

			src.x = CLAMP<double>(src.x, 0, dImgWidth-1);
			src.y = CLAMP<double>(src.y, 0, dImgHeight-1);
			dest.x = CLAMP<double>(dest.x, 0, dImgWidth-1);
			dest.y = CLAMP<double>(dest.y, 0, dImgHeight-1);

			// color interpolation
			ColorInterPolate(&src, &dest, dT, dSrcImgMap, dDstImgMap, &interColor, dImgWidth);

			dMorphMap[i*dImgWidth+j].r = interColor.r;
			dMorphMap[i*dImgWidth+j].g = interColor.g;
			dMorphMap[i*dImgWidth+j].b = interColor.b;
			dMorphMap[i*dImgWidth+j].a = interColor.a;
		}
	}
}

typedef struct Args_struct {
	int i;
	pixel* hMorphMap;
} Args;

void* pthread_imgWrite(void* args);
float stepSize;

int main(int argc,char *argv[]){

    // Timing (https://stackoverflow.com/questions/12231166/timing-algorithm-clock-vs-time-in-c)
    auto start_time_tot = chrono::high_resolution_clock::now();

	// Setup //////////////////

    // Timing
    auto start_time_load = chrono::high_resolution_clock::now();

	parse(argc, argv);
	tempFile = outputPath;
	stepSize = 1.0/steps;

	int linesLen;
	SimpleFeatureLine *hSrcLines, *hDstLines;
	loadLines(&hSrcLines, &hDstLines, &linesLen, inputFileLines.c_str());
	printf("Loaded %d lines\n", linesLen);

	pixel** hMorphMapArr = (pixel**) malloc(sizeof(pixel*) * (steps+1));
	SimpleFeatureLine** hMorphLinesArr = (SimpleFeatureLine**) malloc(sizeof(SimpleFeatureLine*)*(steps+1));

	for (int i = 0; i < steps+1; i++) {
		hMorphMapArr[i] = (pixel*) malloc(sizeof(pixel)*imgHeightOrig*imgWidthOrig);
		simpleLineInterpolate(hSrcLines, hDstLines, &(hMorphLinesArr[i]), linesLen, t);
	}

    // Timing
    auto stop_time_load = chrono::high_resolution_clock::now();
    printf("Time spent on loading images & lines: %ld ms\n", chrono::duration_cast<chrono::milliseconds>(stop_time_load-start_time_load).count() );

	///////////////////////////



    // Image morphing /////////

    // Timing
    auto start_time_morph = chrono::high_resolution_clock::now();

	int dImgWidth = imgHeightOrig; // 1024
	int dImgHeight = imgWidthOrig; // 1024

	// Computes a morphed image for each step based on hMorphLinesArr[i].
	// The morphed image is saved in hMorphMapArr[i];
	for (int i = 0; i < steps+1; i++) {
		t = stepSize*i;
		SimpleFeatureLine* hMorphLines = hMorphLinesArr[i];
		pixel* hMorphMap = hMorphMapArr[i];
		float dT = t;

		// Delete these lines and replace with CUDA variables
		SimpleFeatureLine* dSrcLines = hSrcLines;
		SimpleFeatureLine* dDstLines = hDstLines;
		SimpleFeatureLine* dMorphLines = hMorphLines;
		pixel* dSrcImgMap = hSrcImgMap;
		pixel* dDstImgMap = hDstImgMap;
		pixel* dMorphMap = hMorphMap;

		morphKernel(dSrcLines, dDstLines, dMorphLines, dSrcImgMap, dDstImgMap, dMorphMap, linesLen, dImgWidth, dImgHeight, dT);
	}

    // Timing
    auto stop_time_morph = chrono::high_resolution_clock::now();
    printf("Time spent on morphing: %ld ms\n", chrono::duration_cast<chrono::milliseconds>(stop_time_morph-start_time_morph).count() );

    ///////////////////////////



	// Saving image ///////////

    // Timing
    auto start_time_save = chrono::high_resolution_clock::now();

	for (int i = 0; i < steps+1; i++) {
        float t_i = stepSize*i;
        string path = tempFile + "output-" + to_string(t_i) + ".png";
        imgWrite(path, hMorphMapArr[i], imgWidthOrig, imgHeightOrig);
        free(hMorphMapArr[i]);
    	free(hMorphLinesArr[i]);
	}

    // Timing
    auto stop_time_save = chrono::high_resolution_clock::now();
    printf("Time spent on saving files: %ld ms\n", chrono::duration_cast<chrono::milliseconds>(stop_time_save-start_time_save).count() );

    ///////////////////////////



    // Free host side heap-allocated memory
    free(hMorphMapArr);
	free(hMorphLinesArr);
    free(args_arr);
    free(threads);

	// Free the device side heap-allocated memory
    hipFree(dSrcLines);
    hipFree(dDstLines);
    hipFree(dMorphLines);
    hipFree(dSrcImgMap);
    hipFree(dDstImgMap);
    hipFree(dMorphMap);

    // Timing
    auto stop_time_tot = chrono::high_resolution_clock::now();
    printf("Time spent in total: %ld ms\n", chrono::duration_cast<chrono::milliseconds>(stop_time_tot-start_time_tot).count() );

	return 0;
}


void* pthread_imgWrite(void* args) {
    // TODO: Fill in parallelized code
    return NULL;
}
