#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <signal.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

typedef struct pixel_struct {
	unsigned char r;
	unsigned char g;
	unsigned char b;
	unsigned char a;
} pixel;

#define cudaErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//--------------------------------------------------------------------------------------------------
//--------------------------bilinear interpolation--------------------------------------------------
//--------------------------------------------------------------------------------------------------
// TODO 2 b: Change to device function
void bilinear(pixel* Im, float row, float col, pixel* pix, int width, int height) {
	int cm, cn, fm, fn;
	double alpha, beta;

	cm = (int)ceil(row);
	fm = (int)floor(row);
	cn = (int)ceil(col);
	fn = (int)floor(col);
	alpha = ceil(row) - row;
	beta = ceil(col) - col;

	pix->r = (unsigned char)(alpha*beta*Im[fm*width+fn].r
			+ (1-alpha)*beta*Im[cm*width+fn].r
			+ alpha*(1-beta)*Im[fm*width+cn].r
			+ (1-alpha)*(1-beta)*Im[cm*width+cn].r );
	pix->g = (unsigned char)(alpha*beta*Im[fm*width+fn].g
			+ (1-alpha)*beta*Im[cm*width+fn].g
			+ alpha*(1-beta)*Im[fm*width+cn].g
			+ (1-alpha)*(1-beta)*Im[cm*width+cn].g );
	pix->b = (unsigned char)(alpha*beta*Im[fm*width+fn].b
			+ (1-alpha)*beta*Im[cm*width+fn].b
			+ alpha*(1-beta)*Im[fm*width+cn].b
			+ (1-alpha)*(1-beta)*Im[cm*width+cn].b );
	pix->a = 255;
}
//---------------------------------------------------------------------------
// TODO 2 a: Change to kernel
void bilinear_kernel(pixel* d_pixels_in, pixel* d_pixels_out, int in_width, int in_height, int out_width, int out_height) {
	// TODO 2 c - Parallelize the kernel
	for(int i = 0; i < out_height; i++) {
		for(int j = 0; j < out_width; j++) {
			pixel new_pixel;

			float row = i * (in_height-1) / (float)out_height;
			float col = j * (in_width-1) / (float)out_width;

			bilinear(d_pixels_in, row, col, &new_pixel, in_width, in_height);

			d_pixels_out[i*out_width+j] = new_pixel;
		}
	}
}

int main(int argc, char** argv) {
	stbi_set_flip_vertically_on_load(true);
	stbi_flip_vertically_on_write(true);

	int in_width;
	int in_height;

	pixel* h_pixels_in;
	int channels;
	h_pixels_in = (pixel*) stbi_load(argv[1], &in_width, &in_height, &channels, STBI_rgb_alpha);
	if (h_pixels_in == NULL) exit(1);
	printf("Image dimensions: %dx%d\n", in_width, in_height);

	double scale_x = argc > 2 ? atof(argv[2]) : 2;
	double scale_y = argc > 3 ? atof(argv[3]) : 8;

	int out_width = in_width * scale_x;
	int out_height = in_height * scale_y;

	pixel* h_pixels_out = (pixel*) malloc(sizeof(pixel)*out_width*out_height);

	pixel* d_pixels_in;
	pixel* d_pixels_out;
//TODO 1 a - cuda malloc
	hipMalloc(&d_pixels_in, sizeof(pixel)*in_width*in_height);
	hipMalloc(&d_pixels_out, sizeof(pixel)*out_width*out_height);
//TODO END

   	hipEvent_t start_transfer, stop_transfer;
       	hipEventCreate(&start_transfer);
        hipEventCreate(&stop_transfer);
	hipEventRecord(start_transfer);
//TODO 1 b - cuda memcpy
	hipMemcpy(d_pixels_in, h_pixels_in, sizeof(pixel)*in_width*in_height, hipMemcpyHostToDevice);
	hipMemcpy(d_pixels_out, h_pixels_out, sizeof(pixel)*out_width*out_height, hipMemcpyHostToDevice);
//TODO END

// TODO 1 c - block size and grid size. gridSize should depend on the blockSize and output dimensions.
	dim3 blockSize(1,1);
	dim3 gridSize(1,1);
// TODO END

   	hipEvent_t start, stop;
       	hipEventCreate(&start);
        hipEventCreate(&stop);

	hipEventRecord(start);

//TODO 2 a - GPU computation
// Change the function call so that it becomes a kernel call. Change the input and output pixel variables to be device-side instead of host-side.
        bilinear_kernel(h_pixels_in, h_pixels_out, in_width, in_height, out_width, out_height);
// TODO END

	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float spentTime = 0.0;
	hipEventElapsedTime(&spentTime, start, stop);
	printf("Time spent %.3f seconds\n", spentTime/1000);

//TODO 3 a - Copy the device-side data into the host-side variable

// TODO END

	hipEventRecord(stop_transfer);
	hipEventSynchronize(stop_transfer);
	float spentTimeTransfer = 0.0;
	hipEventElapsedTime(&spentTimeTransfer, start_transfer, stop_transfer);
	printf("Time spent including transfer: %.3f seconds\n", spentTimeTransfer/1000);

	// Writes the host-side data to the output file.
	stbi_write_png("output.png", out_width, out_height, STBI_rgb_alpha, h_pixels_out, sizeof(pixel) * out_width);
//TODO 3 b - Free heap-allocated memory on device and host

// TODO END

	return 0;
}
